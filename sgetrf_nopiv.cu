#include <stdio.h>
#include <stdint.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"
#include "hpl-ai.h"


#define A(i, j) *(A + (i) + (j) * lda)
#define dA(i, j) *(dA + (i) + (j) * ldda)

void sgetrf_nopiv(int m, int n, float *A, int lda) {

    int j;
    int nb = 32;
    int jb = nb;
    int ldda = lda;

    float one = 1.0;
    float none = -1.0;

    float* dA=NULL;

    hipblasStatus_t stat;
    hipblasHandle_t handle;

    // Use unblock code.
    if( nb > m || nb > n ) {
        sgetrf2_nopiv(m, n, A, lda);
    } else {
        hipblasCreate(&handle);
        hipblasSetMathMode(handle, HIPBLAS_TENSOR_OP_MATH);
        hipMalloc((void**)&dA, n*ldda*sizeof(float));
        hipblasSetMatrix(n, n, sizeof(float), A, lda, dA, ldda);
        int min_mn = m<n ? m : n;
        for(j=0; j<min_mn; j+=nb) {
            if( min_mn - j < nb ) {
                jb = min_mn - j;
            }

            // Factor panel
            if( j!=0 ) {
                hipblasGetMatrix(m, jb, sizeof(float), &dA(0, j), ldda, &A(0, j), lda);
            }
            sgetrf2_nopiv(m-j, jb, &A(j, j), lda);

            if( j+jb < n ) {
                hipblasSetMatrix(m-j, jb, sizeof(float), &A(j, j), lda, &dA(j, j), ldda);
                hipblasStrsm(handle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_UNIT, jb, n-j-jb, &one, &dA(j, j), ldda, &dA(j, j+jb), ldda);

                //strsm('L', 'L', 'N', 'U', jb, n-j-jb, 1.0, &A(j, j), lda, &A(j, j+jb), lda);

                if( j+jb < m ) {
                    hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m-j-jb, n-j-jb, jb,
                            &none, &dA(j+jb, j), HIP_R_32F, ldda,
                                   &dA(j, j+jb), HIP_R_32F, ldda,
                            &one, &dA(j+jb, j+jb), HIP_R_32F, ldda, HIP_R_32F, CUBLAS_GEMM_DFALT_TENSOR_OP);
                    //hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m-j-jb, n-j-jb, jb,
                    //        &none, &dA(j+jb, j), ldda, &dA(j, j+jb), ldda, &one, &dA(j+jb, j+jb), ldda);
                    //sgemm('N', 'N', m-j-jb, n-j-jb, jb, -1.0, &A(j+jb, j), lda, &A(j, j+jb), lda, 1.0,
                    //     &A(j+jb, j+jb), lda);


                }

            }
        }
        hipFree(dA);
        hipblasDestroy(handle);
    }
    return;
}

void sgetrf2_nopiv(int m, int n, float *A, int lda) {

  int i;

  if (m <= 1 || n == 0) {
    return;
  }

  if (n == 1) {
    for (i = 1; i < m; i++) {
      A(i, 0) /= A(0, 0);
    }
  } else { // Use recursive code

  int n1 = (m > n ? n : m) / 2;
  int n2 = n - n1;

    sgetrf2_nopiv(m, n1, A, lda);

    strsm('L', 'L', 'N', 'U', n1, n2, 1.0, A, lda, &A(0, n1), lda);

    sgemm('N', 'N', m - n1, n2, n1, -1.0, &A(n1, 0), lda, &A(0, n1), lda, 1.0,
          &A(n1, n1), lda);

    sgetrf2_nopiv(m - n1, n2, &A(n1, n1), lda);

  }
  return;
}
